
#include <hip/hip_runtime.h>
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
//
// RUN: %clang -v --target=i386-unknown-linux \
// RUN:   --sysroot=/tmp/no-cuda-there 2>&1 | FileCheck %s -check-prefix NOCUDA
// RUN: %clang -v --target=i386-unknown-linux \
// RUN:   --cuda-path=%S/Inputs/CUDA/usr/local/cuda 2>&1 | FileCheck %s

// CHECK: Found CUDA installation: {{.*}}/Inputs/CUDA/usr/local/cuda
// NOCUDA-NOT: Found CUDA installation:
