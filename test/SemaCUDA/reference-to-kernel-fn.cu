#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -std=c++11 -fsyntax-only -verify %s
// RUN: %clang_cc1 -std=c++11 -fcuda-is-device -fsyntax-only -verify -DDEVICE %s

// Check that we can reference (get a function pointer to) a __global__
// function from the host side, but not the device side.  (We don't yet support
// device-side kernel launches.)

#include "Inputs/hip/hip_runtime.h"

struct Dummy {};

__global__ void kernel() {}
// expected-note@-1 {{declared here}}
#ifdef DEVICE
// expected-note@-3 {{declared here}}
#endif

typedef void (*fn_ptr_t)();

__host__ __device__ fn_ptr_t get_ptr_hd() {
  return kernel;
#ifdef DEVICE
  // expected-error@-2 {{reference to __global__ function}}
#endif
}
__host__ fn_ptr_t get_ptr_h() {
  return kernel;
}
__device__ fn_ptr_t get_ptr_d() {
  return kernel;  // expected-error {{reference to __global__ function}}
}
