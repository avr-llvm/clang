#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -fcxx-exceptions -fcuda-is-device -fsyntax-only -verify %s
// RUN: %clang_cc1 -fcxx-exceptions -fsyntax-only -verify %s

#include "Inputs/hip/hip_runtime.h"

void host() {
  throw NULL;
  try {} catch(void*) {}
}
__device__ void device() {
  throw NULL;
  // expected-error@-1 {{cannot use 'throw' in __device__ function 'device'}}
  try {} catch(void*) {}
  // expected-error@-1 {{cannot use 'try' in __device__ function 'device'}}
}
__global__ void kernel() {
  throw NULL;
  // expected-error@-1 {{cannot use 'throw' in __global__ function 'kernel'}}
  try {} catch(void*) {}
  // expected-error@-1 {{cannot use 'try' in __global__ function 'kernel'}}
}
