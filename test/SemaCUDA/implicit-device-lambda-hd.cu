// RUN: %clang_cc1 -std=c++11 -fcuda-is-device -verify -verify-ignore-unexpected=note \
// RUN:   -S -o /dev/null %s
// RUN: %clang_cc1 -std=c++11 -verify -fsyntax-only -verify-ignore-unexpected=note \
// RUN:   -DHOST -S -o /dev/null %s
#include "Inputs/hip/hip_runtime.h"

__host__ __device__ void hd_fn() {
  auto f1 = [&] {};
  f1(); // implicitly __host__ __device__

  auto f2 = [&] __device__ {};
  f2();
#ifdef HOST
  // expected-error@-2 {{reference to __device__ function}}
#endif

  auto f3 = [&] __host__ {};
  f3();
#ifndef HOST
  // expected-error@-2 {{reference to __host__ function}}
#endif

  auto f4 = [&] __host__ __device__ {};
  f4();
}


