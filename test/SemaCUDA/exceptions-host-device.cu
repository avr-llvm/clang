// RUN: %clang_cc1 -fcxx-exceptions -fcuda-is-device -verify %s -S -o /dev/null
// RUN: %clang_cc1 -fcxx-exceptions -verify -DHOST %s -S -o /dev/null

#include "Inputs/hip/hip_runtime.h"

// Check that it's an error to use 'try' and 'throw' from a __host__ __device__
// function if and only if it's codegen'ed for device.

#ifdef HOST
// expected-no-diagnostics
#endif

__host__ __device__ void hd1() {
  throw NULL;
  try {} catch(void*) {}
#ifndef HOST
  // expected-error@-3 {{cannot use 'throw' in __host__ __device__ function 'hd1'}}
  // expected-error@-3 {{cannot use 'try' in __host__ __device__ function 'hd1'}}
#endif
}

// No error, never instantiated on device.
inline __host__ __device__ void hd2() {
  throw NULL;
  try {} catch(void*) {}
}
void call_hd2() { hd2(); }

// Error, instantiated on device.
inline __host__ __device__ void hd3() {
  throw NULL;
  try {} catch(void*) {}
#ifndef HOST
  // expected-error@-3 {{cannot use 'throw' in __host__ __device__ function 'hd3'}}
  // expected-error@-3 {{cannot use 'try' in __host__ __device__ function 'hd3'}}
#endif
}
__device__ void call_hd3() { hd3(); }
