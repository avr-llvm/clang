#include "hip/hip_runtime.h"
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// Make sure we handle target overloads correctly.
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu \
// RUN:     -fcuda-target-overloads -emit-llvm -o - %s \
// RUN:     | FileCheck -check-prefix=CHECK-BOTH -check-prefix=CHECK-HOST %s
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fcuda-is-device \
// RUN:     -fcuda-target-overloads -emit-llvm -o - %s \
// RUN:     | FileCheck -check-prefix=CHECK-BOTH -check-prefix=CHECK-DEVICE \
// RUN:       -check-prefix=CHECK-DEVICE-STRICT %s

// Check target overloads handling with disabled call target checks.
// RUN: %clang_cc1 -DNOCHECKS -triple x86_64-unknown-linux-gnu -emit-llvm \
// RUN:    -fcuda-disable-target-call-checks -fcuda-target-overloads -o - %s \
// RUN:     | FileCheck -check-prefix=CHECK-BOTH -check-prefix=CHECK-HOST \
// RUN:    -check-prefix=CHECK-BOTH-NC -check-prefix=CHECK-HOST-NC %s
// RUN: %clang_cc1 -DNOCHECKS -triple nvptx64-nvidia-cuda -emit-llvm \
// RUN:    -fcuda-disable-target-call-checks -fcuda-target-overloads \
// RUN:    -fcuda-is-device -o - %s \
// RUN:     | FileCheck -check-prefix=CHECK-BOTH -check-prefix=CHECK-DEVICE \
// RUN:    -check-prefix=CHECK-BOTH-NC -check-prefix=CHECK-DEVICE-NC %s

#include "Inputs/hip/hip_runtime.h"

typedef int (*fp_t)(void);
typedef void (*gp_t)(void);

// CHECK-HOST: @hp = global i32 ()* @_Z1hv
// CHECK-HOST: @chp = global i32 ()* @ch
// CHECK-HOST: @dhp = global i32 ()* @_Z2dhv
// CHECK-HOST: @cdhp = global i32 ()* @cdh
// CHECK-HOST: @gp = global void ()* @_Z1gv

// CHECK-BOTH-LABEL: define i32 @_Z2dhv()
__device__ int dh(void) { return 1; }
// CHECK-DEVICE: ret i32 1
__host__ int dh(void) { return 2; }
// CHECK-HOST:   ret i32 2

// CHECK-BOTH-LABEL: define i32 @_Z2hdv()
__host__ __device__ int hd(void) { return 3; }
// CHECK-BOTH:   ret i32 3

// CHECK-DEVICE-LABEL: define i32 @_Z1dv()
__device__ int d(void) { return 8; }
// CHECK-DEVICE:   ret i32 8

// CHECK-HOST-LABEL: define i32 @_Z1hv()
__host__ int h(void) { return 9; }
// CHECK-HOST:   ret i32 9

// CHECK-BOTH-LABEL: define void @_Z1gv()
__global__ void g(void) {}
// CHECK-BOTH:   ret void

// mangled names of extern "C" __host__ __device__ functions clash
// with those of their __host__/__device__ counterparts, so
// overloading of extern "C" functions can only happen for __host__
// and __device__ functions -- we never codegen them in the same
// compilation and therefore mangled name conflict is not a problem.

// CHECK-BOTH-LABEL: define i32 @cdh()
extern "C" __device__ int cdh(void) {return 10;}
// CHECK-DEVICE:   ret i32 10
extern "C" __host__ int cdh(void) {return 11;}
// CHECK-HOST:     ret i32 11

// CHECK-DEVICE-LABEL: define i32 @cd()
extern "C" __device__ int cd(void) {return 12;}
// CHECK-DEVICE:   ret i32 12

// CHECK-HOST-LABEL: define i32 @ch()
extern "C" __host__ int ch(void) {return 13;}
// CHECK-HOST:     ret i32 13

// CHECK-BOTH-LABEL: define i32 @chd()
extern "C" __host__ __device__ int chd(void) {return 14;}
// CHECK-BOTH:     ret i32 14

// HD functions are sometimes allowed to call H or D functions -- this
// is an artifact of the source-to-source splitting performed by nvcc
// that we need to mimic. During device mode compilation in nvcc, host
// functions aren't present at all, so don't participate in
// overloading. But in clang, H and D functions are present in both
// compilation modes. Clang normally uses the target attribute as a
// tiebreaker between overloads with otherwise identical priority, but
// in order to match nvcc's behavior, we sometimes need to wholly
// discard overloads that would not be present during compilation
// under nvcc.

template <typename T> T template_vs_function(T arg) { return 15; }
__device__ float template_vs_function(float arg) { return 16; }

// Here we expect to call the templated function during host
// compilation, even if -fcuda-disable-target-call-checks is passed,
// and even though C++ overload rules prefer the non-templated
// function.
// CHECK-BOTH-LABEL: define void @_Z5hd_tfv()
__host__ __device__ void hd_tf(void) {
  template_vs_function(1.0f);
  // CHECK-HOST: call float @_Z20template_vs_functionIfET_S0_(float
  // CHECK-DEVICE: call float @_Z20template_vs_functionf(float
  template_vs_function(2.0);
  // CHECK-HOST: call double @_Z20template_vs_functionIdET_S0_(double
  // CHECK-DEVICE: call float @_Z20template_vs_functionf(float
}

// Calls from __host__ and __device__ functions should always call the
// overloaded function that matches their mode.
// CHECK-HOST-LABEL: define void @_Z4h_tfv()
__host__ void h_tf() {
  template_vs_function(1.0f);
  // CHECK-HOST: call float @_Z20template_vs_functionIfET_S0_(float
  template_vs_function(2.0);
  // CHECK-HOST: call double @_Z20template_vs_functionIdET_S0_(double
}

// CHECK-DEVICE-LABEL: define void @_Z4d_tfv()
__device__ void d_tf() {
  template_vs_function(1.0f);
  // CHECK-DEVICE: call float @_Z20template_vs_functionf(float
  template_vs_function(2.0);
  // CHECK-DEVICE: call float @_Z20template_vs_functionf(float
}

// In case we have a mix of HD and H-only or D-only candidates in the
// overload set, normal C++ overload resolution rules apply first.
template <typename T> T template_vs_hd_function(T arg) { return 15; }
__host__ __device__ float template_vs_hd_function(float arg) { return 16; }

// CHECK-BOTH-LABEL: define void @_Z7hd_thdfv()
__host__ __device__ void hd_thdf() {
  template_vs_hd_function(1.0f);
  // CHECK-HOST: call float @_Z23template_vs_hd_functionf(float
  // CHECK-DEVICE: call float @_Z23template_vs_hd_functionf(float
  template_vs_hd_function(1);
  // CHECK-HOST: call i32 @_Z23template_vs_hd_functionIiET_S0_(i32
  // CHECK-DEVICE-STRICT: call float @_Z23template_vs_hd_functionf(float
  // CHECK-DEVICE-NC: call i32 @_Z23template_vs_hd_functionIiET_S0_(i32
}

// CHECK-HOST-LABEL: define void @_Z6h_thdfv()
__host__ void h_thdf() {
  template_vs_hd_function(1.0f);
  // CHECK-HOST: call float @_Z23template_vs_hd_functionf(float
  template_vs_hd_function(1);
  // CHECK-HOST: call i32 @_Z23template_vs_hd_functionIiET_S0_(i32
}

// CHECK-DEVICE-LABEL: define void @_Z6d_thdfv()
__device__ void d_thdf() {
  template_vs_hd_function(1.0f);
  // CHECK-DEVICE: call float @_Z23template_vs_hd_functionf(float
  template_vs_hd_function(1);
  // Host-only function template is not callable with strict call checks,
  // so for device side HD function will be the only choice.
  // CHECK-DEVICE: call float @_Z23template_vs_hd_functionf(float
}

// Check that overloads still work the same way on both host and
// device side when the overload set contains only functions from one
// side of compilation.
__device__ float device_only_function(int arg) { return 17; }
__device__ float device_only_function(float arg) { return 18; }

__host__ float host_only_function(int arg) { return 19; }
__host__ float host_only_function(float arg) { return 20; }

// CHECK-BOTH-LABEL: define void @_Z6hd_dofv()
__host__ __device__ void hd_dof() {
#ifdef NOCHECKS
  device_only_function(1.0f);
  // CHECK-BOTH-NC: call float @_Z20device_only_functionf(float
  device_only_function(1);
  // CHECK-BOTH-NC: call float @_Z20device_only_functioni(i32
  host_only_function(1.0f);
  // CHECK-BOTH-NC: call float @_Z18host_only_functionf(float
  host_only_function(1);
  // CHECK-BOTH-NC: call float @_Z18host_only_functioni(i32
#endif
}


// CHECK-HOST-LABEL: define void @_Z5hostfv()
__host__ void hostf(void) {
  fp_t hp = h; // CHECK-HOST: store {{.*}} @_Z1hv, {{.*}} %hp,
  fp_t chp = ch; // CHECK-HOST: store {{.*}} @ch, {{.*}} %chp,
  fp_t dhp = dh; // CHECK-HOST: store {{.*}} @_Z2dhv, {{.*}} %dhp,
  fp_t cdhp = cdh; // CHECK-HOST: store {{.*}} @cdh, {{.*}} %cdhp,
  fp_t hdp = hd; // CHECK-HOST: store {{.*}} @_Z2hdv, {{.*}} %hdp,
  fp_t chdp = chd; // CHECK-HOST: store {{.*}} @chd, {{.*}} %chdp,
  gp_t gp = g; // CHECK-HOST: store {{.*}} @_Z1gv, {{.*}} %gp,

  h();     // CHECK-HOST: call i32 @_Z1hv()
  ch();    // CHECK-HOST: call i32 @ch()
  dh();    // CHECK-HOST: call i32 @_Z2dhv()
  cdh();   // CHECK-HOST: call i32 @cdh()
  g<<<0,0>>>();  // CHECK-HOST: call void @_Z1gv()
}

// CHECK-DEVICE-LABEL: define void @_Z7devicefv()
__device__ void devicef(void) {
  fp_t dp = d;   // CHECK-DEVICE: store {{.*}} @_Z1dv, {{.*}} %dp,
  fp_t cdp = cd; // CHECK-DEVICE: store {{.*}} @cd, {{.*}} %cdp,
  fp_t dhp = dh; // CHECK-DEVICE: store {{.*}} @_Z2dhv, {{.*}} %dhp,
  fp_t cdhp = cdh; // CHECK-DEVICE: store {{.*}} @cdh, {{.*}} %cdhp,
  fp_t hdp = hd; // CHECK-DEVICE: store {{.*}} @_Z2hdv, {{.*}} %hdp,
  fp_t chdp = chd; // CHECK-DEVICE: store {{.*}} @chd, {{.*}} %chdp,

  d();     // CHECK-DEVICE: call i32 @_Z1dv()
  cd();    // CHECK-DEVICE: call i32 @cd()
  dh();    // CHECK-DEVICE: call i32 @_Z2dhv()
  cdh();   // CHECK-DEVICE: call i32 @cdh()
}

// CHECK-BOTH-LABEL: define void @_Z11hostdevicefv()
__host__ __device__ void hostdevicef(void) {
#if defined (NOCHECKS)
  fp_t dp = d;   // CHECK-BOTH-NC: store {{.*}} @_Z1dv, {{.*}} %dp,
  fp_t cdp = cd; // CHECK-BOTH-NC: store {{.*}} @cd, {{.*}} %cdp,
  fp_t hp = h; // CHECK-BOTH-NC: store {{.*}} @_Z1hv, {{.*}} %hp,
  fp_t chp = ch; // CHECK-BOTH-NC: store {{.*}} @ch, {{.*}} %chp,
#endif
  fp_t dhp = dh; // CHECK-BOTH: store {{.*}} @_Z2dhv, {{.*}} %dhp,
  fp_t cdhp = cdh; // CHECK-BOTH: store {{.*}} @cdh, {{.*}} %cdhp,
  fp_t hdp = hd; // CHECK-BOTH: store {{.*}} @_Z2hdv, {{.*}} %hdp,
  fp_t chdp = chd; // CHECK-BOTH: store {{.*}} @chd, {{.*}} %chdp,
#if defined (NOCHECKS) && !defined(__CUDA_ARCH__)
  gp_t gp = g; // CHECK-HOST-NC: store {{.*}} @_Z1gv, {{.*}} %gp,
#endif

#if defined (NOCHECKS)
  d();     // CHECK-BOTH-NC: call i32 @_Z1dv()
  cd();    // CHECK-BOTH-NC: call i32 @cd()
  h();     // CHECK-BOTH-NC: call i32 @_Z1hv()
  ch();    // CHECK-BOTH-NC: call i32 @ch()
#endif
  dh();    // CHECK-BOTH: call i32 @_Z2dhv()
  cdh();   // CHECK-BOTH: call i32 @cdh()
#if defined (NOCHECKS) && !defined(__CUDA_ARCH__)
  g<<<0,0>>>();  // CHECK-HOST-NC: call void @_Z1gv()
#endif
}

// Test for address of overloaded function resolution in the global context.
fp_t hp = h;
fp_t chp = ch;
fp_t dhp = dh;
fp_t cdhp = cdh;
gp_t gp = g;

int x;
// Check constructors/destructors for D/H functions
struct s_cd_dh {
  __host__ s_cd_dh() { x = 11; }
  __device__ s_cd_dh() { x = 12; }
  __host__ ~s_cd_dh() { x = 21; }
  __device__ ~s_cd_dh() { x = 22; }
};

struct s_cd_hd {
  __host__ __device__ s_cd_hd() { x = 31; }
  __host__ __device__ ~s_cd_hd() { x = 32; }
};

// CHECK-BOTH: define void @_Z7wrapperv
#if defined(__CUDA_ARCH__)
__device__
#else
__host__
#endif
void wrapper() {
  s_cd_dh scddh;
  // CHECK-BOTH: call void @_ZN7s_cd_dhC1Ev(
  s_cd_hd scdhd;
  // CHECK-BOTH: call void @_ZN7s_cd_hdC1Ev

  // CHECK-BOTH: call void @_ZN7s_cd_hdD1Ev(
  // CHECK-BOTH: call void @_ZN7s_cd_dhD1Ev(
}
// CHECK-BOTH: ret void

// Now it's time to check what's been generated for the methods we used.

// CHECK-BOTH: define linkonce_odr void @_ZN7s_cd_dhC2Ev(
// CHECK-HOST:   store i32 11,
// CHECK-DEVICE: store i32 12,
// CHECK-BOTH: ret void

// CHECK-BOTH: define linkonce_odr void @_ZN7s_cd_hdC2Ev(
// CHECK-BOTH:   store i32 31,
// CHECK-BOTH: ret void

// CHECK-BOTH: define linkonce_odr void @_ZN7s_cd_hdD2Ev(
// CHECK-BOTH: store i32 32,
// CHECK-BOTH: ret void

// CHECK-BOTH: define linkonce_odr void @_ZN7s_cd_dhD2Ev(
// CHECK-HOST:   store i32 21,
// CHECK-DEVICE: store i32 22,
// CHECK-BOTH: ret void

